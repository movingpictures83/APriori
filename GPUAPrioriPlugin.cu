#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "GPUAPrioriPlugin.h"

//factorial function
long int factorial(int x){
	int count = x;
	while (count > 1){
		x = x * (count - 1);
		count--;
	}
	if(x == 0){
		x = 1;
	}	
	return x;	
}

//combinatorics function
long int nCr(int n, int r){
	int y;
	int z;
	int w = n - 1;
	int init = n;
	int x;
	if(r > (n-r)){
	y = r;	
	}
	else{
		y = (n-r);
	}

	z = n - y;
	while(z > 1){
		n = n * w;
		w--;
		z--;
	}
	if( r > (init - r)){
		x = n/factorial(init - r);
	}
	else{
		x = n/factorial(r);
	}
	
	return  x;

}


void GPUAPrioriPlugin::input(std::string file) {
	char val;
	fPointer = fopen(file.c_str(), "r"); 
   	fscanf(fPointer, "%c", &val);
   	while(!feof(fPointer)){
        	if(val == '\n'){
            		size++;
        	}
       		fscanf(fPointer, "%c", &val);
    	}
    	fclose(fPointer);

    	fPointer = fopen(file.c_str(), "r");
   	fscanf(fPointer, "%d", &temp);
	
	//Traverses through each transaction in order to find the max value.
    	while(!feof(fPointer)){
        	fscanf(fPointer, "%d", &temp);
        	if(max < temp){
            		max = temp;
        	}
    	}	
   	fclose(fPointer);

	printf("DATA FILE PARSED\n");
	printf("============================================\n");	
	printf("Total number of transactions found: %d\n", size);
	printf("Maximum number of unique items found: %d\n", max+1);
	printf("============================================\n");	
	printf("APRIORI IMPLEMENTATION BEGINS\n");
	//Creation of table
	cTable = (char*)malloc(sizeof(char) * (max + 1) * size); //Allocates an array of characters for each transaction	
	
	for(i=0; i < (max+1)*size; i++) {
	//	memset(cTable[i], '\0', sizeof(char) * (max + 1) * size); //Initialize all values to 0.
		cTable[i] = '\0';
	}

    	char line[400];
    	char *cNum;
    	fPointer = fopen(file.c_str(), "r");
	for(i = 0; i < size; i++){
		fgets(line, 400, fPointer);

        	cNum = strtok(line, " \n");
        	
		while(cNum != NULL){
            		num = atoi(cNum);
            		cTable[i * (max + 1) + num] = '1';
            		cNum = strtok(NULL, " \n");
        	}	
    	}
}

void GPUAPrioriPlugin::run() {
	//Creating copy of transaction table in the video card memmory
	hipMalloc(&gpuT, size * (max + 1) * sizeof(char));
	hipMemcpy(gpuT, cTable, (size * (max + 1) * sizeof(char)), hipMemcpyHostToDevice);

	//Creates a frequency table of item sets with a Cardinality of 1; where the array index represents the item 
	//number. All the items have their counts initially set to zero
	fTable = (int *)malloc((max + 1) * (cardinality + 1) * sizeof(int));
	for(i = 0; i < max + 1; i++){
		fTable[i * (cardinality + 1)] = i;
		fTable[(i * (cardinality + 1)) + cardinality] = 0;
	}

	int* gpuF;
	hipMalloc(&gpuF, (max + 1) * (cardinality + 1) * sizeof(int));
	hipMemcpy(gpuF, fTable, (max + 1) * (cardinality + 1) * sizeof(int), hipMemcpyHostToDevice);

	//setting the number of cores to be used by the gpu
	numBlocks = (max + 1);
	if(numBlocks > MAX_NUMBER_BLOCKS){
		numBlocks = MAX_NUMBER_BLOCKS;
	}
	counting<<< numBlocks, THREADS_PER_BLOCK>>>(gpuF, gpuT, size, (max + 1), (max + 1),  cardinality);
	
	//setting the number of cores to be used by the gpu
	numBlocks = (max + 1) * (cardinality + 1)/ THREADS_PER_BLOCK + 1;
	if(numBlocks > MAX_NUMBER_BLOCKS){
		numBlocks = MAX_NUMBER_BLOCKS;
	}
	validSets<<< numBlocks, THREADS_PER_BLOCK>>>(gpuF, cardinality, max + 1, mSupport);
	hipMemcpy(fTable, gpuF, ((max + 1) * (cardinality + 1) * sizeof(int)), hipMemcpyDeviceToHost);
	hipFree(gpuF);

	//invalidating elements that are below the support count and counting the remaining eligible elements
	count = 0;
	for(i = 0; i < (max + 1); i++){
		if (fTable[i * (cardinality + 1) + cardinality] != 0){
			count++;
		}
	}


}

void GPUAPrioriPlugin::output(std::string file) {
	//creating new table consisting of only valid items
        int iTable[count];
        j = 0;
        for(i = 0; i < (max + 1); i++){
                if (fTable[i * (cardinality + 1) + cardinality] != 0){
                        iTable[j] = fTable[i * (cardinality + 1)];			
                        j++;
                }
        }
	//creating a tabel to hold the current valid items item and their the a variable for the count of the count
	int * vTable = iTable;
	int lastCount = count;

	while(count > 1){
		cardinality++;

		//temporary array that will hold the new item sets		
		int temp[nCr(count, cardinality) * (cardinality + 1)];

		//array of previous items sets
		int oldSets[nCr(lastCount, cardinality - 1) * cardinality];

		//array that hold one old item set for insertion into table
		int oldEntry[cardinality - 1];

                //function populates old  item set
                k = 0;
                if(cardinality - 1 <= lastCount){
                        for(i = 0; (oldEntry[i] = i) < cardinality - 2; i++); 
                        for(i = 0; i < cardinality - 1; i++){
                                oldSets[(k * cardinality) + i] = vTable[oldEntry[i]];
                        }
                        k++;
                        for(;;){
                                for( i = cardinality - 2; i >= 0 && oldEntry[i] == (lastCount - (cardinality - 1) + i); i--);
                                if(i < 0){
                                        break;
                                }
                                else{
                                        oldEntry[i]++;
                                        for(++i; i < cardinality - 1; i++){
                                                oldEntry[i] = oldEntry[i - 1] + 1;
                                        }
                                        for(j = 0; j < cardinality - 1; j++){
                                                oldSets[(k * cardinality) + j] = vTable[oldEntry[j]];
                                        }
                                        k++;
                                }
                        }
                }

                for(i = 0; i < nCr(lastCount, cardinality - 1); i++){
                        oldSets[(i * cardinality) + cardinality - 1] = 0;
                }

		//array that will hold the information for a single item set before it is added to the 
		//array of all item sets
		int entry[cardinality];

		//function populates new item set
		k = 0;
		if(cardinality <= count){
			for(i = 0; (entry[i] = i) < cardinality - 1; i++);			
			for(i = 0; i < cardinality; i++){
				temp[(k*(cardinality + 1)) + i] = vTable[entry[i]];
			}
			k++;
			for(;;){
				for( i = cardinality - 1; i >= 0 && entry[i] == (count - cardinality + i); i--);
				if(i < 0){
					break;
				}
				else{
					entry[i]++;
					for(++i; i < cardinality; i++){
						entry[i] = entry[i - 1] + 1;
					}
					for(j = 0; j < cardinality; j++){
						temp[(k*(cardinality + 1)) + j] = vTable[entry[j]];
					}
					k++;
				}
			}
		}


						      
		for(i = 0; i < nCr(count, cardinality); i++){
			temp[(i*(cardinality + 1)) + cardinality ] = 0;
		}

		//counting the amount of instances of the item sets amongst the transactions
		int * gpuSet;
		hipMalloc(&gpuSet, sizeof(int) * (cardinality + 1) * nCr(count, cardinality));
		hipMemcpy(gpuSet, temp, sizeof(int) * (cardinality + 1) * nCr(count, cardinality), hipMemcpyHostToDevice);
		numBlocks = nCr(count, cardinality);
		if(numBlocks > MAX_NUMBER_BLOCKS){
			numBlocks = MAX_NUMBER_BLOCKS;
		}
		counting<<< numBlocks, THREADS_PER_BLOCK>>>(gpuSet, gpuT, size, max + 1, nCr(count, cardinality), cardinality);
		hipMemcpy(temp, gpuSet, sizeof(int) * (cardinality + 1) * nCr(count, cardinality), hipMemcpyDeviceToHost);
		hipFree(gpuSet);
		
                //counting the amount of instances of the item sets amongst the transactions
		hipMalloc(&gpuSet, sizeof(int) * cardinality * nCr(lastCount, cardinality - 1));
		hipMemcpy(gpuSet, oldSets, sizeof(int) * cardinality * nCr(lastCount, cardinality - 1), hipMemcpyHostToDevice);
		numBlocks = nCr(lastCount, cardinality - 1);
		if(numBlocks > MAX_NUMBER_BLOCKS){
			numBlocks = MAX_NUMBER_BLOCKS;
		}
		counting<<< numBlocks, THREADS_PER_BLOCK>>>(gpuSet, gpuT, size, max + 1, nCr(lastCount, cardinality - 1), cardinality - 1);
		hipMemcpy(oldSets, gpuSet, sizeof(int) * cardinality * nCr(lastCount, cardinality - 1), hipMemcpyDeviceToHost);
		hipFree(gpuSet);

		//invalidating elements that are below the support count and counting the remaining eligible elements
        	int tCount = count;
		lastCount = tCount;
		count = 0;
        	for(i = 0; i < nCr(tCount, cardinality); i++){
                	if (temp[(i*(cardinality + 1)) + cardinality] < mSupport){
                        	temp[(i * (cardinality + 1)) + cardinality] = 0;
                	}	
                	else{
                        	count++;
                	}
        	}		

		//set Table of valid items
		char valid[max + 1];
		for(i = 0; i <= max; i++){
			valid[i] = '\0';
		}

		for(i = 0; i < nCr(tCount, cardinality); i++){
			for(j = 0; j < cardinality; j++){
				if(temp[(i * (cardinality + 1)) + cardinality] > 0){
					valid[temp[(i * (cardinality + 1)) + j]] = '1';
				}
			}
		}

        	//creating new table consisting of only valid items
        	int rTable[count];
		count = 0;
        	j = 0;
        	for(i = 0; i <= max; i++){
                	if (valid[i] == '1'){
                        	rTable[j] = i;
                        	j++;
				count++;
	                }
        	}	
		vTable = rTable;

		if(count == 0){
			printf("\n=============== MOST FREQUENT SUBSETS ================\n");
	   
	        	for(i = 0; i < nCr(lastCount, cardinality - 1); i++){
				if(oldSets[(i * cardinality) + (cardinality-1)] > mSupport){
                                        printf("Set: {");
                                }
               			for(j = 0; j < cardinality; j++){
					if(oldSets[(i * cardinality) + (cardinality-1)] > mSupport){
                               			if(j == cardinality - 1){
							printf("} Count: %d\n", oldSets[(i * cardinality) + j]);
						}
						else{
							printf("'%d'", oldSets[(i * cardinality) + j]);
						}
                       		 	}	
               		 	}        
			}
			printf("\n");	
		}
	}

}

PluginProxy<GPUAPrioriPlugin> GPUAPrioriPluginProxy = PluginProxy<GPUAPrioriPlugin>("GPUAPriori", PluginManager::getInstance());
